#include "hip/hip_runtime.h"
#include "gpu_jacobi.h"
#include "cpu_jacobi.h"
#include "init.h"
#include "transfer3d_gpu.h"
#include "alloc3d_gpu.h"
#include "alloc3d.h"

#include <hip/hip_runtime.h>

extern "C" {
    #include <stdio.h>
    #include <stdlib.h>
    #ifdef _OPENMP
    #include <omp.h>
    #endif
}

#define DEVICE_0 0
#define DEVICE_1 1

#define BLOCK_SIZE 16

int main(int argc, char *argv[]) {
    int N = 100;
    int iter = 0;
    int iter_max = 100;
    double start_T = 0;
    int gpu_run = 0;

    N = atoi(argv[1]);    // grid size
    iter_max = atoi(argv[2]);  // max. no. of iterations
    start_T = atof(argv[3]);  // start T for all inner grid points
    gpu_run = atof(argv[4]);  // start T for all inner grid points

    double ***u_old = NULL;
    double ***u = NULL;
    double ***f = NULL;

    double delta = 2.0 / (N - 2);

    u_old = d_malloc_3d(N, N, N);
    u = d_malloc_3d(N, N, N);
    f = d_malloc_3d(N, N, N);

    u_init_jac(u_old, N, start_T);
    u_init_jac(u, N, start_T);
    f_init_jac(f, N);

    double start_time, end_time;

    if (!gpu_run) { // CPU run
        start_time = omp_get_wtime();
        cpu_jacobi(u, u_old, f, N, delta, iter_max, &iter);
        end_time = omp_get_wtime();
        printf("CPU %d: iterations done: %d time: %f\n", gpu_run, iter, end_time - start_time);

        free(u_old);
        free(u);
        free(f);

        return 0;
    }

    double ***u_old_gpu = NULL;
    double ***u_gpu = NULL;
    double ***f_gpu = NULL;

    u_old_gpu = d_malloc_3d_gpu(N, N, N);
    u_gpu = d_malloc_3d_gpu(N, N, N);
    f_gpu = d_malloc_3d_gpu(N, N, N);

    transfer_3d(u_old_gpu, u_old, N, N, N, hipMemcpyHostToDevice);
    transfer_3d(u_gpu, u, N, N, N, hipMemcpyHostToDevice);
    transfer_3d(f_gpu, f, N, N, N, hipMemcpyHostToDevice);

    switch (gpu_run) { // GPU run
        case 0:
            return 0;

        case 1:
            hipSetDevice(DEVICE_0);
            start_time = omp_get_wtime();
            run_gpu_jacobi_1(u_gpu, u_old_gpu, f_gpu, N, delta, iter_max, &iter);
            end_time = omp_get_wtime();
            printf("GPU %d: iterations done: %d time: %f\n", gpu_run, iter, end_time - start_time);
            break;

        case 2:
            hipSetDevice(DEVICE_0);
            double *u_old_1d_gpu = NULL;
            double *u_1d_gpu = NULL;
            double *f_1d_gpu = NULL;

            int size = N * N * N * sizeof(double);

            hipMalloc((void**)&u_old_1d_gpu, size_f);
            hipMalloc((void**)&u_1d_gpu, size_f);
            hipMalloc((void**)&f_1d_gpu, size_f);

            transfer_3d_to_1d(u_old_1d_gpu, u_old_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_to_1d(u_1d_gpu, u_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_to_1d(f_1d_gpu, f_gpu, N, N, N, hipMemcpyDeviceToDevice);

            dim3 dim_grid = dim3(N, N, N);
            dim3 dim_block = dim3(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);

            start_time = omp_get_wtime();
            run_gpu_jacobi_2(u_1d_gpu, u_old_1d_gpu, f_1d_gpu, N, delta, iter_max, &iter, dim_grid, dim_block);
            end_time = omp_get_wtime();
            printf("GPU %d: iterations done: %d time: %f\n", gpu_run, iter, end_time - start_time);

            transfer_3d_from_1d(u_old_gpu, u_old_1d_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_from_1d(u_gpu, u_1d_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_from_1d(f_gpu, f_1d_gpu, N, N, N, hipMemcpyDeviceToDevice);

            hipFree(u_old_1d_gpu);
            hipFree(u_1d_gpu);
            hipFree(f_1d_gpu);
            break;
    }

    free_gpu(u_old_gpu);
    free_gpu(u_gpu);
    free_gpu(f_gpu);

    return 0;
}
