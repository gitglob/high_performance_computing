#include "hip/hip_runtime.h"
#include "gpu_jacobi.h"
#include "cpu_jacobi.h"
#include "init.h"
#include "transfer3d_gpu.h"
#include "alloc3d_gpu.h"
#include "alloc3d.h"

#include <hip/hip_runtime.h>

extern "C" {
    #include <stdio.h>
    #include <stdlib.h>
    #ifdef _OPENMP
    #include <omp.h>
    #endif
}

#define DEVICE_0 0
#define DEVICE_1 1

#define BLOCK_SIZE 16

int main(int argc, char *argv[]) {
    int N = 100;
    int iter = 0;
    int iter_max = 100;
    double start_T = 0;
    int gpu_run = 0;

    N = atoi(argv[1]);    // grid size
    iter_max = atoi(argv[2]);  // max. no. of iterations
    start_T = atof(argv[3]);  // start T for all inner grid points
    gpu_run = atof(argv[4]);  // start T for all inner grid points

    double ***u_old = NULL;
    double ***u = NULL;
    double ***f = NULL;

    double delta = 2.0 / (N - 2);

    u_old = d_malloc_3d(N, N, N);
    u = d_malloc_3d(N, N, N);
    f = d_malloc_3d(N, N, N);

    u_init_jac(u_old, N, start_T);
    u_init_jac(u, N, start_T);
    f_init_jac(f, N);

    double start_time, end_time;

    if (!gpu_run) { // CPU run
        start_time = omp_get_wtime();
        cpu_jacobi(u, u_old, f, N, delta, iter_max, &iter);
        end_time = omp_get_wtime();
        printf("CPU %d: iterations done: %d time: %f\n", gpu_run, iter, end_time - start_time);

        free(u_old);
        free(u);
        free(f);

        return 0;
    }

    double ***u_old_gpu = NULL;
    double ***u_gpu = NULL;
    double ***f_gpu = NULL;

    hipSetDevice(DEVICE_0);

    printf("Allocation 3d on gpu\n");
    u_old_gpu = d_malloc_3d_gpu(N, N, N);
    u_gpu = d_malloc_3d_gpu(N, N, N);
    f_gpu = d_malloc_3d_gpu(N, N, N);

    printf("Transferring to host\n");
    transfer_3d(u_old_gpu, u_old, N, N, N, hipMemcpyHostToDevice);
    transfer_3d(u_gpu, u, N, N, N, hipMemcpyHostToDevice);
    transfer_3d(f_gpu, f, N, N, N, hipMemcpyHostToDevice);

    switch (gpu_run) { // GPU run
        case 0:
            return 0;

        case 1:
            printf("Case 1\n");
            start_time = omp_get_wtime();
            run_gpu_jacobi_1(u_gpu, u_old_gpu, f_gpu, N, delta, iter_max, &iter);
            end_time = omp_get_wtime();
            printf("GPU %d: iterations done: %d time: %f\n", gpu_run, iter, end_time - start_time);

        case 2:
            printf("Case 2\n");
            hipSetDevice(DEVICE_0);
            double *u_old_1d_gpu = NULL;
            double *u_1d_gpu = NULL;
            double *f_1d_gpu = NULL;

            transfer_3d_to_1d(u_old_1d_gpu, u_old_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_to_1d(u_1d_gpu, u_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_to_1d(f_1d_gpu, f_gpu, N, N, N, hipMemcpyDeviceToDevice);

            dim3 dim_grid = dim3(N, N, N);
            dim3 dim_block = dim3(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);

            start_time = omp_get_wtime();
            run_gpu_jacobi_2(u_1d_gpu, u_old_1d_gpu, f_1d_gpu, N, delta, iter_max, &iter, dim_grid, dim_block);
            end_time = omp_get_wtime();
            printf("GPU %d: iterations done: %d time: %f\n", gpu_run, iter, end_time - start_time);

            transfer_3d_from_1d(u_old_gpu, u_old_1d_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_from_1d(u_gpu, u_1d_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_from_1d(f_gpu, f_1d_gpu, N, N, N, hipMemcpyDeviceToDevice);

            hipFree(u_old_1d_gpu);
            hipFree(u_1d_gpu);
            hipFree(f_1d_gpu);
    }

    free_gpu(u_old_gpu);
    free_gpu(u_gpu);
    free_gpu(f_gpu);

    return 0;
}
