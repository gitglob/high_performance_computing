#include "hip/hip_runtime.h"
#include "gpu_jacobi.h"
#include "cpu_jacobi.h"
#include "init.h"
#include "transfer3d_gpu.h"
#include "alloc3d_gpu.h"
#include "alloc3d.h"
#include "print.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern "C" {
#include <stdio.h>
#include <stdlib.h>
#ifdef _OPENMP
#include <omp.h>
#endif
}

#define DEVICE_0 0
#define DEVICE_1 1

#define BLOCK_SIZE 8

int main(int argc, char *argv[]) {
    int N = 0;
    int iter = 0;
    int iter_max = 100;
    double start_T = 0;
    int gpu_run = 0;
    int output_type = 0;
    double tolerance;

    N = atoi(argv[1]);    // grid size
    iter_max = atoi(argv[2]);  // max. no. of iterations
    start_T = atof(argv[3]);  // start T for all inner grid points
    gpu_run = atof(argv[4]); // 0 -> run CPU, 1/2/3-> run on GPU

    if (gpu_run == 3){
        N = 512; // this is hardcoded, it can probably be done another way
    }

    output_type = atof(argv[5]); // 3=.bin, 4=.vtk
    tolerance = atof(argv[6]);

    double ***u_old = NULL;
    double ***u = NULL;
    double ***f = NULL;

    double delta = 2.0 / (N - 2);

    u_old = d_malloc_3d(N, N, N);
    u = d_malloc_3d(N, N, N);
    f = d_malloc_3d(N, N, N);

    u_init_jac(u_old, N, start_T);
    u_init_jac(u, N, start_T);
    f_init_jac(f, N);

    double start_time, end_time;

    if (!gpu_run) { // CPU run
        start_time = omp_get_wtime();
        cpu_jacobi(u, u_old, f, N, delta, iter_max, &iter);
        end_time = omp_get_wtime();
        printf("CPU %d: iterations done: %d time: %f\n", gpu_run, iter, end_time - start_time);

        free(u_old);
        free(u);
        free(f);

        return 0;
    }

    double ***u_old_gpu = NULL;
    double ***u_gpu = NULL;
    double ***f_gpu = NULL;

    hipSetDevice(DEVICE_0);
    u_old_gpu = d_malloc_3d_gpu(N, N, N);
    u_gpu = d_malloc_3d_gpu(N, N, N);
    f_gpu = d_malloc_3d_gpu(N, N, N);

    transfer_3d(u_old_gpu, u_old, N, N, N, hipMemcpyHostToDevice);
    transfer_3d(u_gpu, u, N, N, N, hipMemcpyHostToDevice);
    transfer_3d(f_gpu, f, N, N, N, hipMemcpyHostToDevice);

    int grid_size = (int) N / BLOCK_SIZE;
    if (N % BLOCK_SIZE > 0) grid_size++;

    dim3 dim_grid;
    dim3 dim_block;
    if (gpu_run!=3){
      dim_grid = dim3(grid_size, grid_size, grid_size);
      dim_block = dim3(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
    }

    double *u_old_1d_gpu = NULL;
    double *u_1d_gpu = NULL;
    double *f_1d_gpu = NULL;

    switch (gpu_run) { // GPU run
        case 0: {
            return 0;
        };
            break;

        case 1: {
            hipSetDevice(DEVICE_0);
            start_time = omp_get_wtime();
            run_gpu_jacobi_1(u_gpu, u_old_gpu, f_gpu, N, delta, iter_max, &iter);
            end_time = omp_get_wtime();
            printf("GPU %d: iterations done: %d time: %f\n", gpu_run, iter, end_time - start_time);
        };

            break;

        case 2: {
            hipSetDevice(DEVICE_0);

            int size = N * N * N * sizeof(double);

            hipMalloc((void **) &u_old_1d_gpu, size);
            hipMalloc((void **) &u_1d_gpu, size);
            hipMalloc((void **) &f_1d_gpu, size);

            transfer_3d_to_1d(u_old_1d_gpu, u_old_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_to_1d(u_1d_gpu, u_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_to_1d(f_1d_gpu, f_gpu, N, N, N, hipMemcpyDeviceToDevice);

            start_time = omp_get_wtime();
            run_gpu_jacobi_2(u_1d_gpu, u_old_1d_gpu, f_1d_gpu, N, delta, iter_max, &iter, dim_grid, dim_block);
            end_time = omp_get_wtime();
            printf("GPU %d: iterations done: %d time: %f\n", gpu_run, iter, end_time - start_time);

            transfer_3d_from_1d(u_old_gpu, u_old_1d_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_from_1d(u_gpu, u_1d_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_from_1d(f_gpu, f_1d_gpu, N, N, N, hipMemcpyDeviceToDevice);

            hipFree(u_old_1d_gpu);
            hipFree(u_1d_gpu);
            hipFree(f_1d_gpu);
        };
            break;

        case 3: {
            // define size
            int size = N * N * N * sizeof(double);

            // create 2 sub-matrices in host
            double ***u0_old = NULL;
            double ***u0 = NULL;
            double ***f0 = NULL;
            double ***u1_old = NULL;
            double ***u1 = NULL;
            double ***f1 = NULL;
            u0_old = d_malloc_3d(N / 2, N, N);
            u0 = d_malloc_3d(N / 2, N, N);
            f0 = d_malloc_3d(N / 2, N, N);
            u1_old = d_malloc_3d(N / 2, N, N);
            u1 = d_malloc_3d(N / 2, N, N);
            f1 = d_malloc_3d(N / 2, N, N);
            for (int i = 0; i < N; i++) {
                for (int j = 0; j < N; j++) {
                    for (int k = 0; k < N; k++) {
                        if (i < N / 2) {
                            u0_old[i][j][k] = u[i][j][k];
                            u0[i][j][k] = u[i][j][k];
                            f0[i][j][k] = u[i][j][k];
                        }
                        else {
                            u1_old[i - (N / 2)][j][k] = u[i][j][k];
                            u1[i - (N / 2)][j][k] = u[i][j][k];
                            f1[i - (N / 2)][j][k] = u[i][j][k];
                        }
                    }
                }
            }

            // transfer matrices to devices
            double ***u0_old_gpu = NULL;
            double ***u0_gpu = NULL;
            double ***f0_gpu = NULL;
            hipSetDevice(DEVICE_0);
            u0_old_gpu = d_malloc_3d_gpu(N / 2, N, N);
            u0_gpu = d_malloc_3d_gpu(N / 2, N, N);
            f0_gpu = d_malloc_3d_gpu(N / 2, N, N);
            transfer_3d(u0_old_gpu, u0_old, N / 2, N, N, hipMemcpyHostToDevice);
            transfer_3d(u0_gpu, u0, N / 2, N, N, hipMemcpyHostToDevice);
            transfer_3d(f0_gpu, f0, N / 2, N, N, hipMemcpyHostToDevice);
            double ***u1_old_gpu = NULL;
            double ***u1_gpu = NULL;
            double ***f1_gpu = NULL;
            hipSetDevice(DEVICE_1);
            u1_old_gpu = d_malloc_3d_gpu(N / 2, N, N);
            u1_gpu = d_malloc_3d_gpu(N / 2, N, N);
            f1_gpu = d_malloc_3d_gpu(N / 2, N, N);
            transfer_3d(u1_old_gpu, u1_old, N / 2, N, N, hipMemcpyHostToDevice);
            transfer_3d(u1_gpu, u1, N / 2, N, N, hipMemcpyHostToDevice);
            transfer_3d(f1_gpu, f1, N / 2, N, N, hipMemcpyHostToDevice);
            checkCudaErrors(hipDeviceSynchronize());

            // convert matrices to 1d
            hipSetDevice(DEVICE_0);
            double *u0_old_1d_gpu = NULL;
            double *u0_1d_gpu = NULL;
            double *f0_1d_gpu = NULL;
            hipMalloc((void **) &u0_old_1d_gpu, size / 2);
            hipMalloc((void **) &u0_1d_gpu, size / 2);
            hipMalloc((void **) &f0_1d_gpu, size / 2);
            transfer_3d_to_1d(u0_old_1d_gpu, u0_old_gpu, N / 2, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_to_1d(u0_1d_gpu, u0_gpu, N / 2, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_to_1d(f0_1d_gpu, f0_gpu, N / 2, N, N, hipMemcpyDeviceToDevice);
            hipSetDevice(DEVICE_1);
            double *u1_1d_gpu = NULL;
            double *u1_old_1d_gpu = NULL;
            double *f1_1d_gpu = NULL;
            hipMalloc((void **) &u1_old_1d_gpu, size / 2);
            hipMalloc((void **) &u1_1d_gpu, size / 2);
            hipMalloc((void **) &f1_1d_gpu, size / 2);
            transfer_3d_to_1d(u1_old_1d_gpu, u1_old_gpu, N / 2, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_to_1d(u1_1d_gpu, u1_gpu, N / 2, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_to_1d(f1_1d_gpu, f1_gpu, N / 2, N, N, hipMemcpyDeviceToDevice);

            hipSetDevice(DEVICE_0);
            hipDeviceEnablePeerAccess(1, 0);
            hipSetDevice(DEVICE_1);
            hipDeviceEnablePeerAccess(0, 0);

            checkCudaErrors(hipDeviceSynchronize());

            printf("N: %d, Grid size: %d, Block size: %d\n", N, N / BLOCK_SIZE, BLOCK_SIZE);
            dim_grid = dim3(N / (BLOCK_SIZE*2), N / BLOCK_SIZE, N / BLOCK_SIZE);
            dim_block = dim3(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
            start_time = omp_get_wtime();
            run_gpu_jacobi_3(u0_1d_gpu, u0_old_1d_gpu, f0_1d_gpu, N, delta, iter_max, &iter, dim_grid, dim_block,
                             u1_1d_gpu, u1_old_1d_gpu, f1_1d_gpu);
            end_time = omp_get_wtime();
            printf("2 GPUs (%d): iterations done: %d time: %f\n", gpu_run, iter, end_time - start_time);
            checkCudaErrors(hipDeviceSynchronize());

            // return u0 to host
            hipSetDevice(DEVICE_0);
            double *h_u0_1d_gpu = NULL;
            hipHostMalloc((void **) &h_u0_1d_gpu, size / 2);
            hipMemcpy(h_u0_1d_gpu, u0_1d_gpu, size / 2, hipMemcpyDeviceToHost);
            checkCudaErrors(hipDeviceSynchronize());

            // return u1 to host
            hipSetDevice(DEVICE_1);
            double *h_u1_1d_gpu = NULL;
            hipHostMalloc((void **) &h_u1_1d_gpu, size / 2);
            hipMemcpy(h_u1_1d_gpu, u1_1d_gpu, size / 2, hipMemcpyDeviceToHost);
            checkCudaErrors(hipDeviceSynchronize());

            // combine matrices in host
            double *h_u_1d_gpu = NULL;
            hipHostMalloc((void **) &h_u_1d_gpu, size);
            hipMemcpy(h_u_1d_gpu, h_u0_1d_gpu, size/2, hipMemcpyHostToHost);
            checkCudaErrors(hipDeviceSynchronize());
            hipMemcpy(h_u_1d_gpu + (N * N * N / 2), h_u1_1d_gpu, size/2, hipMemcpyHostToHost);
            checkCudaErrors(hipDeviceSynchronize());

            // transfer u in host back to 3d
            transfer_3d_from_1d(u, h_u_1d_gpu, N, N, N, hipMemcpyHostToHost);
            checkCudaErrors(hipDeviceSynchronize());

            /* debug
            int sum3 =0;
            for (int i=0; i<N; i++){
              for (int j=0; j<N; j++){
                for (int k=0; k<N; k++){
                  if (u[i][j][k]<2){// && u[i][j][k]!=20){
                    sum3++;
                  }
                }
              }
            }
            printf("h_u 3D : %d %d\n",sum3,N*N*N);
            */

            hipSetDevice(DEVICE_0);
            hipFree(u0_old_1d_gpu);
            hipFree(u0_1d_gpu);
            hipFree(f0_1d_gpu);
            hipSetDevice(DEVICE_1);
            hipFree(u1_old_1d_gpu);
            hipFree(u1_1d_gpu);
            hipFree(f1_1d_gpu);

            hipHostFree(h_u0_1d_gpu);
            hipHostFree(h_u1_1d_gpu);
            hipHostFree(h_u_1d_gpu);

            hipDeviceDisablePeerAccess(DEVICE_0);
            hipDeviceDisablePeerAccess(DEVICE_1);
        };
            break;

        case 4: {
            hipSetDevice(DEVICE_0);

            int size = N * N * N * sizeof(double);

            hipMalloc((void **) &u_old_1d_gpu, size);
            hipMalloc((void **) &u_1d_gpu, size);
            hipMalloc((void **) &f_1d_gpu, size);

            transfer_3d_to_1d(u_old_1d_gpu, u_old_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_to_1d(u_1d_gpu, u_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_to_1d(f_1d_gpu, f_gpu, N, N, N, hipMemcpyDeviceToDevice);

            start_time = omp_get_wtime();
            run_gpu_jacobi_4(u_1d_gpu, u_old_1d_gpu, f_1d_gpu, N, delta, iter_max, &iter, dim_grid, dim_block,
                             &tolerance);
            end_time = omp_get_wtime();
            printf("GPU %d: iterations done: %d time: %f, tolerance: %f\n", gpu_run, iter, end_time - start_time,
                   tolerance);

            transfer_3d_from_1d(u_old_gpu, u_old_1d_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_from_1d(u_gpu, u_1d_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_from_1d(f_gpu, f_1d_gpu, N, N, N, hipMemcpyDeviceToDevice);

            hipFree(u_old_1d_gpu);
            hipFree(u_1d_gpu);
            hipFree(f_1d_gpu);
        };
            break;

        case 5: {
            hipSetDevice(DEVICE_0);

            int size = N * N * N * sizeof(double);

            hipMalloc((void **) &u_old_1d_gpu, size);
            hipMalloc((void **) &u_1d_gpu, size);
            hipMalloc((void **) &f_1d_gpu, size);

            transfer_3d_to_1d(u_old_1d_gpu, u_old_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_to_1d(u_1d_gpu, u_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_to_1d(f_1d_gpu, f_gpu, N, N, N, hipMemcpyDeviceToDevice);

            start_time = omp_get_wtime();
            run_gpu_jacobi_5(u_1d_gpu, u_old_1d_gpu, f_1d_gpu, N, delta, iter_max, &iter, dim_grid, dim_block,
                             &tolerance);
            end_time = omp_get_wtime();
            printf("GPU %d: iterations done: %d time: %f, tolerance: %f\n", gpu_run, iter, end_time - start_time,
                   tolerance);

            transfer_3d_from_1d(u_old_gpu, u_old_1d_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_from_1d(u_gpu, u_1d_gpu, N, N, N, hipMemcpyDeviceToDevice);
            transfer_3d_from_1d(f_gpu, f_1d_gpu, N, N, N, hipMemcpyDeviceToDevice);

            hipFree(u_old_1d_gpu);
            hipFree(u_1d_gpu);
            hipFree(f_1d_gpu);
        };
            break;
    }

    if (gpu_run!=3){
      transfer_3d(u, u_gpu, N, N, N, hipMemcpyDeviceToHost);
    }

    char *output_prefix = "poisson3_res";
    char *output_ext = "";
    char output_filename[FILENAME_MAX];

    switch (output_type) {
        case 0:
            // no output at all
            break;
        case 3:
            output_ext = ".bin";
            sprintf(output_filename, "%s_%d%s", output_prefix, N, output_ext);
            fprintf(stderr, "Write binary dump to %s: ", output_filename);
            print_binary(output_filename, N, u);
            break;
        case 4:
            output_ext = ".vtk";
            sprintf(output_filename, "%s_%d%s", output_prefix, N, output_ext);
            fprintf(stderr, "Write VTK file to %s: ", output_filename);
            print_vtk(output_filename, N, u);
            break;
        default:
            fprintf(stderr, "Non-supported output type!\n");
            break;
    }

    free_gpu(u_old_gpu);
    free_gpu(u_gpu);
    free_gpu(f_gpu);
    free(u);

    return 0;
}
