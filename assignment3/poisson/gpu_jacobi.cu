#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

#define BLOCK_SIZE 8

__global__
void gpu_jacobi_1(double ***u, double ***u_old, double ***f, int N, double ***temp_pointer, int delta_2, double div_val) {
    int i, j, k;
    for (i = 1; i < N - 1; ++i) {
        for (j = 1; j < N - 1; ++j) {
            for (k = 1; k < N - 1; ++k) {
                u[i][j][k] =  (u_old[i - 1][j][k] + u_old[i + 1][j][k]
                         + u_old[i][j - 1][k] + u_old[i][j + 1][k]
                         + u_old[i][j][k - 1] + u_old[i][j][k + 1]
                         + delta_2 * f[i][j][k]) * div_val;
            }
        }
    }

    temp_pointer = u;
    u = u_old;
    u_old = temp_pointer;
}

void run_gpu_jacobi_1(double ***u, double ***u_old, double ***f, int N, int delta, int iter_max, int *iter) {

    double delta_2 = delta * delta;
    double div_val = 1.0 / 6.0;
    double ***temp_pointer;

    while (*iter < iter_max) {
        gpu_jacobi_1<<<1, 1>>>(u, u_old, f, N, temp_pointer, delta_2, div_val);
        checkCudaErrors(hipDeviceSynchronize());
        (*iter)++;
    }
}

__global__
void gpu_jacobi_2(double *u, double *u_old, double *f, int N, int delta_2, double div_val) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

//    printf("x: %d, y: %d, z: %d\n", x,y,z);
    if (x > 0 && x < N - 1 && y > 0 && y < N - 1 && z > 0 && z < N - 1) {
            u[N * N * x + N * y + z] = (u_old[N * N * (x - 1) + N * y + z] + u_old[N * N * (x + 1) + N * y + z]
                                      + u_old[N * N * x + N * (y - 1) + z] + u_old[N * N * x + N * (y + 1) + z]
                                      + u_old[N * N * x + N * y + (z - 1)] + u_old[N * N * x + N * y + (z + 1)]
                                      + delta_2 * f[N * N * x + N * y + z]) * div_val;
    }


}

void run_gpu_jacobi_2(double *u, double *u_old, double *f, int N, int delta, int iter_max, int *iter, dim3 dim_grid, dim3 dim_block) {

    double delta_2 = delta * delta;
    double div_val = 1.0 / 6.0;
    double *temp_pointer = NULL;

    //hipMalloc((void**)&temp_pointer, N*N*N);

    while (*iter < iter_max) {
        gpu_jacobi_2<<<dim_grid, dim_block>>>(u, u_old, f, N, delta_2, div_val);
        checkCudaErrors(hipDeviceSynchronize());
        (*iter)++;
        temp_pointer = u;
        u = u_old;
        u_old = temp_pointer;
    }
}

__global__
void gpu_jacobi_3(double *u, double *u_old, double *f, int N, double *temp_pointer, int delta_2, double div_val,
                  double *u_, double *u_old_, double *f_) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if ( x == N/2 ) { // check if we are at the border for the second half
        printf("2nd half border!!\n");
        u[N * N * x + N * y + z] = (u_old_[N * N * (x - 1) + N * y + z] + u_old[N * N * (x + 1) + N * y + z]
                                  + u_old[N * N * x + N * (y - 1) + z] + u_old[N * N * x + N * (y + 1) + z]
                                  + u_old[N * N * x + N * y + (z - 1)] + u_old[N * N * x + N * y + (z + 1)]
                                  + delta_2 * f[N * N * x + N * y + z]) * div_val;
    }
    else if (x == ((N/2) -1)){
        printf("1st half border!!\n");
        u[N * N * x + N * y + z] = (u_old[N * N * (x - 1) + N * y + z] + u_old_[N * N * (x + 1) + N * y + z]
                                  + u_old[N * N * x + N * (y - 1) + z] + u_old[N * N * x + N * (y + 1) + z]
                                  + u_old[N * N * x + N * y + (z - 1)] + u_old[N * N * x + N * y + (z + 1)]
                                  + delta_2 * f[N * N * x + N * y + z]) * div_val;
    }
    else if (x > 1 && x < N - 1 && y > 1 && y < N - 1 && z > 1 && z < N - 1) {
        u[N * N * x + N * y + z] = (u_old[N * N * (x - 1) + N * y + z] + u_old[N * N * (x + 1) + N * y + z]
                                  + u_old[N * N * x + N * (y - 1) + z] + u_old[N * N * x + N * (y + 1) + z]
                                  + u_old[N * N * x + N * y + (z - 1)] + u_old[N * N * x + N * y + (z + 1)]
                                  + delta_2 * f[N * N * x + N * y + z]) * div_val;
    }

    temp_pointer = u;
    u = u_old;
    u_old = temp_pointer;
}

void run_gpu_jacobi_3(double *u, double *u_old, double *f, int N, int delta, int iter_max, int *iter, dim3 dim_grid, dim3 dim_block,
                      double *u_, double *u_old_, double *f_) {

    double delta_2 = delta * delta;
    double div_val = 1.0 / 6.0;
    double *temp_pointer;

    while (*iter < iter_max) {
        gpu_jacobi_3<<<dim_grid, dim_block>>>(u, u_old, f, N, temp_pointer, delta_2, div_val,
                                              u_, u_old_, f_);
        checkCudaErrors(hipDeviceSynchronize());
        (*iter)++;
    }
}


__global__
void gpu_jacobi_4(double *u, double *u_old, double *f, int N, int delta_2, double div_val, double *d) {

    double value = 0;
    double norm_diff, norm;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x > 0 && x < N - 1 && y > 0 && y < N - 1 && z > 0 && z < N - 1) {
            u[N * N * x + N * y + z] = (u_old[N * N * (x - 1) + N * y + z] + u_old[N * N * (x + 1) + N * y + z]
                                      + u_old[N * N * x + N * (y - 1) + z] + u_old[N * N * x + N * (y + 1) + z]
                                      + u_old[N * N * x + N * y + (z - 1)] + u_old[N * N * x + N * y + (z + 1)]
                                      + delta_2 * f[N * N * x + N * y + z]) * div_val;

            norm_diff = u[N * N * x + N * y + z] - u_old[N * N * x + N * y + z];
            norm = norm_diff * norm_diff;

            atomicAdd(d, norm);
    }

}

void run_gpu_jacobi_4(double *u, double *u_old, double *f, int N, int delta, int iter_max, int *iter, dim3 dim_grid, dim3 dim_block, double *tolerance) {
    double d = 100000;
    double delta_2 = delta * delta;
    double div_val = 1.0 / 6.0;
    double *temp_pointer = NULL;

    double *d_gpu;
    hipMalloc((void**)&d_gpu, sizeof(double));

    printf("Iter: %i\n", *iter);
    printf("Tolerance: %f\n", *tolerance);
    while (d > *tolerance && *iter < iter_max) {
        d = 0;
        hipMemcpy(d_gpu, &d, sizeof(double), hipMemcpyHostToDevice);
        gpu_jacobi_4<<<dim_grid, dim_block>>>(u, u_old, f, N, delta_2, div_val, d_gpu);
        hipMemcpy(&d, d_gpu, sizeof(double), hipMemcpyDeviceToHost);
        checkCudaErrors(hipDeviceSynchronize());
        d = sqrt(d);
        // printf("d: %f\n", d);
        temp_pointer = u;
        u = u_old;
        u_old = temp_pointer;
        (*iter)++;
    }

    *tolerance = d;
    hipFree(d_gpu);
}


__inline__ __device__
double warpReduceSum(double value) {
    for(int i = 16; i > 0; i/=2)
        value += __shfl_down_sync(-1, value, i);
    return value;
}

__inline__ __device__
double blockReduceSum(double value) {
    __shared__ double smem[32]; // Max 32 warp sums
    int blockThreadIdx = (threadIdx.x + (threadIdx.y * BLOCK_SIZE) + (threadIdx.z * BLOCK_SIZE *BLOCK_SIZE));

    if (blockThreadIdx < 32)
        smem[blockThreadIdx] = 0;

    __syncthreads();
    value = warpReduceSum(value);

    if (blockThreadIdx % 32 == 0)
        smem[blockThreadIdx / warpSize] = value;

    __syncthreads();

    if (blockThreadIdx < 32)
        value = smem[blockThreadIdx];

    return warpReduceSum(value);
}

// __global__
// void reduction_presum(double *a, int n, double *res) {
//     int idx = threadIdx.x + blockIdx.x * blockDim.x;
//     double value = 0;
//     for (int i = idx; i < n; i += blockDim.x * gridDim.x)
//         value += a[i];

//     value = blockReduceSum(value);
//     if (threadIdx.x == 0)
//         atomicAdd(res, value);
// }

__global__
void gpu_jacobi_5(double *u, double *u_old, double *f, int N, int delta_2, double div_val, double *d) {

    double norm_diff, norm;

    norm = 0;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x > 0 && x < N - 1 && y > 0 && y < N - 1 && z > 0 && z < N - 1) {
            u[N * N * x + N * y + z] = (u_old[N * N * (x - 1) + N * y + z] + u_old[N * N * (x + 1) + N * y + z]
                                      + u_old[N * N * x + N * (y - 1) + z] + u_old[N * N * x + N * (y + 1) + z]
                                      + u_old[N * N * x + N * y + (z - 1)] + u_old[N * N * x + N * y + (z + 1)]
                                      + delta_2 * f[N * N * x + N * y + z]) * div_val;

            norm_diff = u[N * N * x + N * y + z] - u_old[N * N * x + N * y + z];
            
            norm = norm_diff * norm_diff;  
    }
    norm = blockReduceSum(norm);
    if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
            atomicAdd(d, norm);
}

void run_gpu_jacobi_5(double *u, double *u_old, double *f, int N, int delta, int iter_max, int *iter, dim3 dim_grid, dim3 dim_block, double *tolerance) {
    double d = 100000;
    double delta_2 = delta * delta;
    double div_val = 1.0 / 6.0;
    double *temp_pointer = NULL;

    double *d_gpu;
    hipMalloc((void**)&d_gpu, sizeof(double));

    printf("Iter: %i\n", *iter);
    printf("Tolerance: %f\n", *tolerance);
    while (d > *tolerance && *iter < iter_max) {
        d = 0;
        hipMemcpy(d_gpu, &d, sizeof(double), hipMemcpyHostToDevice);
        gpu_jacobi_4<<<dim_grid, dim_block>>>(u, u_old, f, N, delta_2, div_val, d_gpu);
        hipMemcpy(&d, d_gpu, sizeof(double), hipMemcpyDeviceToHost);
        checkCudaErrors(hipDeviceSynchronize());
        d = sqrt(d);
        // printf("d: %f\n", d);
        temp_pointer = u;
        u = u_old;
        u_old = temp_pointer;
        (*iter)++;
    }

    *tolerance = d;
    hipFree(d_gpu);
}