#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

__global__
void gpu_jacobi_1(double ***u, double ***u_old, double ***f, int N, double *temp_pointer, int delta_2, double div_val) {
    int i, j, k;
    for (i = 1; i < N - 1; ++i) {
        for (j = 1; j < N - 1; ++j) {
            for (k = 1; k < N - 1; ++k) {
                u[i][j][k] =  (u_old[i - 1][j][k] + u_old[i + 1][j][k]
                         + u_old[i][j - 1][k] + u_old[i][j + 1][k]
                         + u_old[i][j][k - 1] + u_old[i][j][k + 1]
                         + delta_2 * f[i][j][k]) * div_val;
            }
        }
    }

    temp_pointer = u;
    u = u_old;
    u_old = temp_pointer;
    checkCudaErrors(hipDeviceSynchronize());
}

void run_gpu_jacobi_1(double ***u, double ***u_old, double ***f, int N, int delta, int iter_max, int *iter) {

    double delta_2 = delta * delta;
    double div_val = 1.0 / 6.0;
    double *temp_pointer;

    while (*iter < iter_max) {
        gpu_jacobi_1_thread<<<1, 1>>>(u, u_old, f, N, temp_pointer, delta_2, div_val);
        iter++;
    }
}

__global__
void gpu_jacobi_2(double *u, double *u_old, double *f, int N, double *temp_pointer, int delta_2, double div_val) {

        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;
        int z = blockIdx.z * blockDim.z + threadIdx.z;

        if (x > 1 && x < N - 1 && y > 1 && y < N - 1 && z > 1 && z < N - 1) {
                u[N * N * x + N * y + z] = (u_old[N * N * (x - 1) + N * y + z] + u_old[N * N * (x + 1) + N * y + z]
                                          + u_old[N * N * x + N * (y - 1) + z] + u_old[N * N * x + N * (y + 1) + z]
                                          + u_old[N * N * x + N * y + (z - 1)] + u_old[N * N * x + N * y + (z + 1)]
                                          + delta_2 * f[N * N * x + N * y + z]) * div_val;
            }

            temp_pointer = u;
            u = u_old;
            u_old = temp_pointer;
            checkCudaErrors(hipDeviceSynchronize());
        }
    }
}

void run_gpu_jacobi_2(double *u, double *u_old, double *f, int N, int delta, int iter_max, int *iter, dim3 dim_grid, dim3 dim_block) {

    double delta_2 = delta * delta;
    double div_val = 1.0 / 6.0;
    double *temp_pointer;

    while (*iter < iter_max) {
        gpu_jacobi_2<<<dim_grid, dim_block>>>(u, u_old, f, N, temp_pointer, delta_2, div_val);
        iter++;
    }
}