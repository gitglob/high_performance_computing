#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

#define BLOCK_SIZE 8

__global__
void gpu_jacobi_1(double ***u, double ***u_old, double ***f, int N, double ***temp_pointer, int delta_2, double div_val) {
    int i, j, k;
    for (i = 1; i < N - 1; ++i) {
        for (j = 1; j < N - 1; ++j) {
            for (k = 1; k < N - 1; ++k) {
                u[i][j][k] =  (u_old[i - 1][j][k] + u_old[i + 1][j][k]
                         + u_old[i][j - 1][k] + u_old[i][j + 1][k]
                         + u_old[i][j][k - 1] + u_old[i][j][k + 1]
                         + delta_2 * f[i][j][k]) * div_val;
            }
        }
    }

    temp_pointer = u;
    u = u_old;
    u_old = temp_pointer;
}

__global__
void gpu_jacobi_2(double *u, double *u_old, double *f, int N, int delta_2, double div_val) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x > 0 && x < N - 1 && y > 0 && y < N - 1 && z > 0 && z < N - 1) {
            u[N * N * x + N * y + z] = (u_old[N * N * (x - 1) + N * y + z] + u_old[N * N * (x + 1) + N * y + z]
                                      + u_old[N * N * x + N * (y - 1) + z] + u_old[N * N * x + N * (y + 1) + z]
                                      + u_old[N * N * x + N * y + (z - 1)] + u_old[N * N * x + N * y + (z + 1)]
                                      + delta_2 * f[N * N * x + N * y + z]) * div_val;
    }


}

void run_gpu_jacobi_2(double *u, double *u_old, double *f, int N, int delta, int iter_max, int *iter, dim3 dim_grid, dim3 dim_block) {

    double delta_2 = delta * delta;
    double div_val = 1.0 / 6.0;
    double *temp_pointer = NULL;

    //hipMalloc((void**)&temp_pointer, N*N*N);

    while (*iter < iter_max) {
        gpu_jacobi_2<<<dim_grid, dim_block>>>(u, u_old, f, N, delta_2, div_val);
        checkCudaErrors(hipDeviceSynchronize());
        (*iter)++;
        temp_pointer = u;
        u = u_old;
        u_old = temp_pointer;
    }
}

__global__
void gpu_jacobi_32(double *u, double *u_old, double *f, int N, int delta_2, double div_val,
                  double *u_old_) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x>0 && y>0 && z>0 && x<(N/2-1) && y<N-1 && z<N-1){ // inside the half cube
        u[N * N * x + N * y + z] = (u_old[N/2 * N * (x-1) + N * y + z] + u_old[N/2 * N * (x + 1) + N * y + z]
                                  + u_old[N/2 * N * x + N * (y - 1) + z] + u_old[N/2 * N * x + N * (y + 1) + z]
                                  + u_old[N/2 * N * x + N * y + (z - 1)] + u_old[N/2 * N * x + N * y + (z + 1)]
                                  + delta_2 * f[N/2 * N * x + N * y + z]) * div_val;
    }
    else if (x == 0 && y>0 && z>0 && y<N-1 && z<N-1) { // border
        u[N * N * x + N * y + z] = (u_old_[N/2 * N * (N/2-1) + N * y + z] + u_old[N/2 * N * (x + 1) + N * y + z]
                                  + u_old[N/2 * N * x + N * (y - 1) + z] + u_old[N/2 * N * x + N * (y + 1) + z]
                                  + u_old[N/2 * N * x + N * y + (z - 1)] + u_old[N/2 * N * x + N * y + (z + 1)]
                                  + delta_2 * f[N/2 * N * x + N * y + z]) * div_val;
    }
}

__global__
void gpu_jacobi_31(double *u, double *u_old, double *f, int N,int delta_2, double div_val,
                  double *u_old_) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x>0 && y>0 && z>0 && x<(N/2-1) && y<N-1 && z<N-1){ // inside the half cube
        u[N * N * x + N * y + z] = (u_old[N/2 * N * (x - 1) + N * y + z] + u_old_[N/2 * N * (x + 1) + N * y + z]
                                  + u_old[N/2 * N * x + N * (y - 1) + z] + u_old[N/2 * N * x + N * (y + 1) + z]
                                  + u_old[N/2 * N * x + N * y + (z - 1)] + u_old[N/2 * N * x + N * y + (z + 1)]
                                  + delta_2 * f[N/2 * N * x + N * y + z]) * div_val;
    }
    else if (x == N/2-1 && y>0 && z>0 && y<N-1 && z<N-1) { // border
        u[N * N * x + N * y + z] = (u_old[N/2 * N * (x - 1) + N * y + z] + u_old_[N/2 * N * (0) + N * y + z]
                                  + u_old[N/2 * N * x + N * (y - 1) + z] + u_old[N/2 * N * x + N * (y + 1) + z]
                                  + u_old[N/2 * N * x + N * y + (z - 1)] + u_old[N/2 * N * x + N * y + (z + 1)]
                                  + delta_2 * f[N/2 * N * x + N * y + z]) * div_val;
    }
}

void run_gpu_jacobi_3(double *u0, double *u0_old, double *f0, int N, int delta, int iter_max, int *iter, dim3 dim_grid, dim3 dim_block,
                      double *u1, double *u1_old, double *f1) {

    double delta_2 = delta * delta;
    double div_val = 1.0 / 6.0;
    double *temp_pointer = NULL;

    while (*iter < iter_max) {
        hipSetDevice(0);
        gpu_jacobi_31<<<dim_grid, dim_block>>>(u0, u0_old, f0, N, delta_2, div_val,
                                              u1_old);
        hipSetDevice(1);
        gpu_jacobi_32<<<dim_grid, dim_block>>>(u1, u1_old, f1, N, delta_2, div_val,
                                              u0_old);
        hipSetDevice(0);
        checkCudaErrors(hipDeviceSynchronize());
        hipSetDevice(1);
        checkCudaErrors(hipDeviceSynchronize());
        (*iter)++;
        temp_pointer = u0;
        u0 = u0_old;
        u0_old = temp_pointer;
        temp_pointer = u1;
        u1 = u1_old;
        u1_old = temp_pointer;
    }
}


__global__
void gpu_jacobi_4(double *u, double *u_old, double *f, int N, int delta_2, double div_val, double *d) {

    double value = 0;
    double norm_diff, norm;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x > 0 && x < N - 1 && y > 0 && y < N - 1 && z > 0 && z < N - 1) {
            u[N * N * x + N * y + z] = (u_old[N * N * (x - 1) + N * y + z] + u_old[N * N * (x + 1) + N * y + z]
                                      + u_old[N * N * x + N * (y - 1) + z] + u_old[N * N * x + N * (y + 1) + z]
                                      + u_old[N * N * x + N * y + (z - 1)] + u_old[N * N * x + N * y + (z + 1)]
                                      + delta_2 * f[N * N * x + N * y + z]) * div_val;

            norm_diff = u[N * N * x + N * y + z] - u_old[N * N * x + N * y + z];
            norm = norm_diff * norm_diff;

            atomicAdd(d, norm);
    }

}

void run_gpu_jacobi_4(double *u, double *u_old, double *f, int N, int delta, int iter_max, int *iter, dim3 dim_grid, dim3 dim_block, double *tolerance) {
    double d = 100000;
    double delta_2 = delta * delta;
    double div_val = 1.0 / 6.0;
    double *temp_pointer = NULL;

    double *d_gpu;
    hipMalloc((void**)&d_gpu, sizeof(double));

    printf("Iter: %i\n", *iter);
    printf("Tolerance: %f\n", *tolerance);
    while (d > *tolerance && *iter < iter_max) {
        d = 0;
        hipMemcpy(d_gpu, &d, sizeof(double), hipMemcpyHostToDevice);
        gpu_jacobi_4<<<dim_grid, dim_block>>>(u, u_old, f, N, delta_2, div_val, d_gpu);
        hipMemcpy(&d, d_gpu, sizeof(double), hipMemcpyDeviceToHost);
        checkCudaErrors(hipDeviceSynchronize());
        d = sqrt(d);
        // printf("d: %f\n", d);
        temp_pointer = u;
        u = u_old;
        u_old = temp_pointer;
        (*iter)++;
    }

    *tolerance = d;
    hipFree(d_gpu);
}


__inline__ __device__
double warpReduceSum(double value) {
    for(int i = 16; i > 0; i/=2)
        value += __shfl_down_sync(-1, value, i);
    return value;
}

__inline__ __device__
double blockReduceSum(double value) {
    __shared__ double smem[32]; // Max 32 warp sums
    int blockThreadIdx = (threadIdx.x + (threadIdx.y * BLOCK_SIZE) + (threadIdx.z * BLOCK_SIZE *BLOCK_SIZE));

    if (blockThreadIdx < 32)
        smem[blockThreadIdx] = 0;

    __syncthreads();
    value = warpReduceSum(value);

    if (blockThreadIdx % 32 == 0)
        smem[blockThreadIdx / warpSize] = value;

    __syncthreads();

    if (blockThreadIdx < 32)
        value = smem[blockThreadIdx];

    return warpReduceSum(value);
}

// __global__
// void reduction_presum(double *a, int n, double *res) {
//     int idx = threadIdx.x + blockIdx.x * blockDim.x;
//     double value = 0;
//     for (int i = idx; i < n; i += blockDim.x * gridDim.x)
//         value += a[i];

//     value = blockReduceSum(value);
//     if (threadIdx.x == 0)
//         atomicAdd(res, value);
// }

__global__
void gpu_jacobi_5(double *u, double *u_old, double *f, int N, int delta_2, double div_val, double *d) {

    double norm_diff, norm;

    norm = 0;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x > 0 && x < N - 1 && y > 0 && y < N - 1 && z > 0 && z < N - 1) {
            u[N * N * x + N * y + z] = (u_old[N * N * (x - 1) + N * y + z] + u_old[N * N * (x + 1) + N * y + z]
                                      + u_old[N * N * x + N * (y - 1) + z] + u_old[N * N * x + N * (y + 1) + z]
                                      + u_old[N * N * x + N * y + (z - 1)] + u_old[N * N * x + N * y + (z + 1)]
                                      + delta_2 * f[N * N * x + N * y + z]) * div_val;

            norm_diff = u[N * N * x + N * y + z] - u_old[N * N * x + N * y + z];

            norm = norm_diff * norm_diff;
    }
    norm = blockReduceSum(norm);
    if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
            atomicAdd(d, norm);
}

void run_gpu_jacobi_5(double *u, double *u_old, double *f, int N, int delta, int iter_max, int *iter, dim3 dim_grid, dim3 dim_block, double *tolerance) {
    double d = 100000;
    double delta_2 = delta * delta;
    double div_val = 1.0 / 6.0;
    double *temp_pointer = NULL;

    double *d_gpu;
    hipMalloc((void**)&d_gpu, sizeof(double));

    printf("Iter: %i\n", *iter);
    printf("Tolerance: %f\n", *tolerance);
    while (d > *tolerance && *iter < iter_max) {
        d = 0;
        hipMemcpy(d_gpu, &d, sizeof(double), hipMemcpyHostToDevice);
        gpu_jacobi_4<<<dim_grid, dim_block>>>(u, u_old, f, N, delta_2, div_val, d_gpu);
        hipMemcpy(&d, d_gpu, sizeof(double), hipMemcpyDeviceToHost);
        checkCudaErrors(hipDeviceSynchronize());
        d = sqrt(d);
        // printf("d: %f\n", d);
        temp_pointer = u;
        u = u_old;
        u_old = temp_pointer;
        (*iter)++;
    }

    *tolerance = d;
    hipFree(d_gpu);
}